#include <iostream>
#include <vector>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"


//CUBLAS matmul

static __inline__ void modify(hipblasHandle_t handle, float (*A)[10000], float (*B)[10000], float (*C)[10000], \
const float a,  const float b, int n, int r, int c)
{
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &a, *A, n, *B, n,\
     &b, *C, n);
}

int main(void) {
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    int M = 10000, N = 10000;

    float (*A)[10000] = new float [N][10000];
    float (*B)[10000] = new float [N][10000];
    float (*C)[10000] = new float [N][10000];

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            A[i][j] = 2.0f + (i*0.01f);
            B[i][j] = 2.0f + (i*0.01f);
            C[i][j] = 6.0f + (i*0.01f);
        }
    }
    // Allocate 3 arrays on GPU
    float (*dA)[10000], (*dB)[10000], (*dC)[10000];
    size_t a = sizeof(float[10000][10000]);
    hipMalloc(&dA, a);
    hipMalloc(&dB, a);
    hipMalloc(&dC, a);
    
    hipMemcpy(dA, A, a,hipMemcpyHostToDevice);
    hipMemcpy(dB, B, a,hipMemcpyHostToDevice);
    hipMemcpy(dC, C, a,hipMemcpyHostToDevice);
    stat = hipblasCreate(&handle);
    modify (handle, dA, dB, dC, 1.0f, 1.0f, N, N, N);
    hipDeviceSynchronize();

    hipMemcpy(C, dC, a,hipMemcpyDeviceToHost);
    
    printf("hello this is test %f, %f", C[1][1], C[22][22]);
    // //Free GPU memory
    hipFree(dA), hipFree(dB), hipFree(dC);
    
    // // Free CPU memory
    free(A), free(B), free(C);
 }